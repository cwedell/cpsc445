
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

__global__ void sqrtcalc(float* inputs, int size) {
  int myrank = blockIdx.x * blockDim.x + threadIdx.x;
  if(myrank < size) {
    float mynum = inputs[myrank];
    mynum = sqrt(mynum);
    inputs[myrank] = mynum;
  }
	__syncthreads();
}

int main() {
	string filein = "input.csv";
	string fileout = "output.csv";
	vector<float> nums;
	string line = "";

	try {
		ifstream instream(filein);
		if(!instream.good()) {
			throw invalid_argument("File does not exist");
		}
		if(instream.is_open()) {
			while(getline(instream, line)) {
				nums.push_back(stof(line));
			}
		}
		instream.close();
	}
	catch(exception& e) {
		cout << "Invalid input" << endl;
	}

	int size = nums.size();
	float* sqrts = &nums[0];
	float* inputs;
	hipMalloc((void**)&inputs, size * sizeof(float));
	hipMemcpy(inputs, sqrts, size * sizeof(float), hipMemcpyHostToDevice);

	sqrtcalc<<<(int)ceil(size/1000), 1000>>>(inputs, size);
	hipDeviceSynchronize();

	float* outputs = new float[size];
	hipMemcpy(outputs, inputs, size * sizeof(float), hipMemcpyDeviceToHost);

	ofstream outstream(fileout);
	for(int i = 0; i < size; ++i) {
		outstream << outputs[i] << endl;
	}
	outstream.close();

	hipFree(inputs);
	free(outputs);
	return 0;
}
