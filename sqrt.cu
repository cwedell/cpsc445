
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

__global__ void sqrtcalc(float* inputs) {
	float mynum = inputs[threadIdx.x];
	mynum = sqrt(mynum);
	inputs[threadIdx.x] = mynum;
	__syncthreads();
}

int main() {
	string filein = "input.csv";
	string fileout = "output.csv";
	vector<float> nums;
	string line = "";

	try {
		ifstream instream(filein);
		if(!instream.good()) {
			throw invalid_argument("File does not exist");
		}
		if(instream.is_open()) {
			while(getline(instream, line)) {
				nums.push_back(stof(line));
			}
		}
		instream.close();
	}
	catch(exception& e) {
		cout << "Invalid input" << endl;
	}

	int size = nums.size();
	float* sqrts = &nums[0];
	float* inputs;
	hipMalloc((void**)&inputs, size * sizeof(float));
	hipMemcpy(inputs, sqrts, size * sizeof(float), hipMemcpyHostToDevice);

	sqrtcalc<<<1, size>>>(inputs);
	hipDeviceSynchronize();

	float* outputs = new float[size];
	hipMemcpy(outputs, inputs, size * sizeof(float), hipMemcpyDeviceToHost);

	ofstream outstream(fileout);
	for(int i = 0; i < size; ++i) {
		outstream << outputs[i] << endl;
    cout << "sqrt of " << nums[i] << " = " << outputs[i] << endl;
	}
	outstream.close();

	hipFree(inputs);
	free(outputs);
	return 0;
}
