
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>

using namespace std;

__global__ void charcount(char* dnainput, int* countA, int* countT, int* countG, int* countC) {
	char* mychar = (char*)dnainput[threadIdx.x];
  // atomicAdd() adds 1 to each variable and prevents race conditions
	if(mychar == (char*)'A') {
		atomicAdd(countA, 1);
	} else if(mychar == (char*)'T') {
		atomicAdd(countT, 1);
	} else if(mychar == (char*)'G') {
		atomicAdd(countG, 1);
	} else if(mychar == (char*)'C') {
		atomicAdd(countC, 1);
	}
	__syncthreads();
}

int main() {
	string filein = "dna.txt";
	string fileout = "output.txt";
	string dna = "";
	string line = "";

	try {
		ifstream instream(filein);
		if(!instream.good()) {
			throw invalid_argument("File does not exist");
		}
		if(instream.is_open()) {
			while(getline(instream, line)) {
				dna = line;
			}
		}
		instream.close();
	}
	catch(exception& e) {
		cout << "Invalid input" << endl;
	}

	char* dnachar = new char[dna.length() + 1];
	strcpy(dnachar, dna.c_str());
	int size = dna.length();

  // send char array to device
	char* dnainput;
	hipMalloc((void**)&dnainput, size * sizeof(char));
	hipMemcpy(dnainput, dnachar, size * sizeof(char), hipMemcpyHostToDevice);

  // send counters to device
	int* countA;
	int* countT;
	int* countG;
	int* countC;
	hipMalloc((void**)&countA, sizeof(int));
	hipMalloc((void**)&countT, sizeof(int));
	hipMalloc((void**)&countG, sizeof(int));
	hipMalloc((void**)&countC, sizeof(int));
	hipMemcpy(countA, 0, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(countT, 0, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(countG, 0, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(countC, 0, sizeof(int), hipMemcpyHostToDevice);

  // one thread per character
	charcount<<<1, size>>>(dnainput, countA, countT, countG, countC);
	hipDeviceSynchronize();

  // get counters from device
	int Acounted;
	int Tcounted;
	int Gcounted;
	int Ccounted;
	hipMemcpy(&Acounted, countA, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&Tcounted, countT, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&Gcounted, countG, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&Ccounted, countC, sizeof(int), hipMemcpyDeviceToHost);

	ofstream outstream(fileout);
	outstream << "A " << Acounted << endl;
	outstream << "T " << Tcounted << endl;
	outstream << "G " << Gcounted << endl;
	outstream << "C " << Ccounted << endl;
	outstream.close();

	hipFree(dnainput);
	free(dnachar);
	return 0;
}
