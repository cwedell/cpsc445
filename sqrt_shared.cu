
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

__global__ void sqrtcalc(float* inputs, int size) {
	int myrank = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ float shinputs[];
	if(myrank < size) {
    shinputs[myrank] = inputs[myrank];
		shinputs[myrank] = sqrt(shinputs[myrank]);
	}
	__syncthreads();
  printf("my rank is %f\n", myrank);
  if(myrank < size) {
		inputs[myrank] = shinputs[myrank];
    if(myrank % 1000 == 0) {
      printf("rank %f is writing %f to %f", myrank, shinputs[myrank], inputs[myrank]);
    }
	}
  __syncthreads();
}

int main() {
	string filein = "input.csv";
	string fileout = "output.csv";
	vector<float> nums;
	string line = "";

	try {
		ifstream instream(filein);
		if(!instream.good()) {
			throw invalid_argument("File does not exist");
		}
		if(instream.is_open()) {
			while(getline(instream, line)) {
				nums.push_back(stof(line));
			}
		}
		instream.close();
	}
	catch(exception& e) {
		cout << "Invalid input" << endl;
	}

	int size = nums.size();
	float* sqrts = &nums[0];
	float* inputs;
	hipMalloc((void**)&inputs, size * sizeof(float));
	hipMemcpy(inputs, sqrts, size * sizeof(float), hipMemcpyHostToDevice);

	sqrtcalc<<<ceil((float)size/1000), 1000, size * sizeof(float)>>>(inputs, size);
	hipDeviceSynchronize();

	float* outputs = new float[size];
	hipMemcpy(outputs, inputs, size * sizeof(float), hipMemcpyDeviceToHost);

	ofstream outstream(fileout);
	for(int i = 0; i < size; ++i) {
		outstream << outputs[i] << endl;
	}
	outstream.close();

	hipFree(inputs);
	free(outputs);
	return 0;
}
